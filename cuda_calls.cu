
#include <hip/hip_runtime.h>

const int BUFFER_SIZE = 1024 * 1024 * 1024;
const int NUM_BLOCKS = 1024;
const int NUM_THREADS = 1024;
const int NUM_STREAMS = 16;

__global__ void kernel(int *buffer, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        buffer[i] = i;
    }
}

void cuda_calls() {
    int *buffer;
    hipMalloc(&buffer, BUFFER_SIZE);

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        kernel<<<NUM_BLOCKS, NUM_THREADS, 0, streams[i]>>>(buffer, BUFFER_SIZE);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(buffer);
}